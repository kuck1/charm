#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

// nvcc 036sgemm.c -lcublas

#define DGEMM dgemm_
#define DSPEV dspev_
#define PRINTF printf
#define EXIT exit
#define CLOCKS_PER_SEC_C  1000000
#define MAXTIME 2147.48

void cputime(double *);
void get_iter_Tmat(double *,double *,int );
void get_diag_Tmat(double *,double *,int );
void get_unit_Tmat(double *,int );

extern "C" { void DGEMM (char *, char *, int *, int *, int *,double *,double *, int *, double *, int *, double *, double *, int * ); }

int matmul(double *X, int *LDX, int *ITYPE_X, 
    double *Y, int *LDY, int *ITYPE_Y,
    double *Z, int *LDZ, int *NRZ, int *NCZ, int *NXY,
    double *ALPHA, double *BETA)
{
    int m = *NRZ;
    int n = *NCZ;
    int k = *NXY;

    //char MATX=(ITYPE_X) ? 'N' : 'T';
    //char MATY=(ITYPE_Y) ? 'N' : 'T';
    // DGEMM(&MATX,&MATY,NRZ,NCZ,NXY,ALPHA,X,LDX,Y,LDY,BETA,Z,LDZ);

    hipblasOperation_t MATX = (ITYPE_X) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t MATY = (ITYPE_Y) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    // cudaError_t cudaStat;           // cudaMalloc status 
    // cublasStatus_t stat;            // CUBLAS functions status 
    hipblasHandle_t handle;          // CUBLAS context

    // Step 1: Allocate memory on the device:
    double *d_X, *d_Y, *d_Z;
    hipMalloc(&d_X, (m*k)*sizeof(double));          // X is an m x k matrix
    hipMalloc(&d_Y, (k*n)*sizeof(double));          // Y is a  k X n matix
    hipMalloc(&d_Z, (m*n)*sizeof(double));          // Z is an m x n matix

    hipblasCreate(&handle);                           // initialize CUBLAS context

    // Step 2: Initailize device memory from host:
    hipblasSetMatrix(m, k, sizeof(double), X, m, d_X, m);       
    hipblasSetMatrix(k, n, sizeof(double), Y, k, d_Y, k); 
    hipblasSetMatrix(m, n, sizeof(double), Z, m, d_Z, m);

    // Step 3: Perform operation, function launches kernel on GPU itself
    hipblasDgemm(handle, MATX, MATY, m, n, k, ALPHA, d_X, m, d_Y, k, BETA, d_Z, m);

    // Step 4: Copy the result back to the host:
    hipblasGetMatrix(m, n, sizeof(double), d_Z, m, Z, m);

    // Step 5: Clean up
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_Z);
    hipblasDestroy(handle);
}

//DGEMM ( TRANSA, TRANSB, M, N, K, ALPHA, A, LDA, B, LDB, BETA, C, LDC )

/* cublasStatus_t cublasDgemm(cublasHandle_t handle, cublasOperation_t transa, cublasOperation_t transb, 
                              int m, int n, int k, const double *alpha, const double *A, int lda, 
                              const double *B, int ldb, const double *beta, double *C, int ldc)
*/

int device_matmul(double *d_X, int *LDX, int *ITYPE_X, 
    double *d_Y, int *LDY, int *ITYPE_Y,
    double *d_Z, int *LDZ, int *NRZ, int *NCZ, int *NXY,
    double *ALPHA, double *BETA, hipblasHandle_t handle)
{
    int m = *NRZ;
    int n = *NCZ;
    int k = *NXY;

    hipblasOperation_t MATX = (ITYPE_X) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t MATY = (ITYPE_Y) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    //cublasHandle_t handle;          // CUBLAS context

    hipblasDgemm(handle, MATX, MATY, m, n, k, ALPHA, d_X, m, d_Y, k, BETA, d_Z, m);
}


#define _USE_LAPACK_

#ifdef _USE_LAPACK_
extern "C" {void   DSPEV(char *, char *, int *, double [], double [], double [], int *, double [], int *);}
#endif
//=======================================================================
//ccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccc
//=======================================================================
  int main()
//=======================================================================
  {// begin routine 
//=======================================================================
// I) Set up the problem

   int nstate;
   PRINTF("\n============================================\n");
   PRINTF("Enter the matrix size : ");scanf("%d",&nstate);
   int nstate_sq = nstate*nstate;

   double *S     = new double[nstate_sq];
   double *Tunit = new double[nstate_sq];
   double *Tdiag = new double[nstate_sq];
   double *Titer = new double[nstate_sq];
   
   PRINTF("Using random input\n\n");
   for(int i=0;i<nstate_sq;i++){S[i]=0.0;}
   for(int i=0;i<nstate;i++){int ind =i+nstate*i;S[ind]=2.0;}

   double seed=14571.0;
   srand48((long) seed);
   for(int i=0;i<nstate;i++){
   for(int j=i;i<nstate;i++){
     int ind  = i+nstate*j;
     int indt = j+nstate*i;
     int n=1,ierr=0;
     double rand=drand48();
     S[ind]  += (rand-0.5)*2.0e-3;
     S[indt] = S[ind];
   }}//endfor

//=======================================================================
// II) Try three methods

//   get_unit_Tmat(Tunit,nstate);
//   get_diag_Tmat(S,Tdiag,nstate);
   get_iter_Tmat(S,Titer,nstate);
   get_iter_Tmat(S,Titer,nstate);
   get_iter_Tmat(S,Titer,nstate);

//=======================================================================
// III) Check the error of the iterative method

   double err=0.0;
   for(int i=0;i<nstate_sq;i++){
     double tmp=Tdiag[i]-Titer[i];    
     tmp = tmp*tmp;
     err = (err > tmp ? err : tmp);
   }//endfor
   err = sqrt(err);
   PRINTF("Maximum error in any element : %g\n",err);

   err=0.0;
   for(int i=0;i<nstate;i++){
   for(int j=i;j<nstate;j++){
     int ind  = i + j*nstate;
     int indt = j + i*nstate;
     double tmp=Titer[ind]-Titer[indt];    
     tmp = tmp*tmp;
     err = (err > tmp ? err : tmp);
   }}//endfor
   err = sqrt(err);
   PRINTF("Deviation from symmetric : %g\n",err);
   PRINTF("============================================\n\n");

//=======================================================================
  }//end routine
//=======================================================================



//============================================================================
//cccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccc
//============================================================================
// Diagonalize S and construct T=S^{-1/2} using eigenvalues and eigenvectors
//============================================================================

void get_diag_Tmat(double *S,double *T,int nstate)

//============================================================================
  {//begin routine
//============================================================================
// I) Get some scratch

   double cpu1,cpu2;
   cputime(&cpu1);

   int nstate_sq     = nstate*nstate;
   double *umat      = new double[nstate_sq];
   double *scr_mat1  = new double[nstate_sq];
   double *scr_mat2  = new double[nstate_sq];
   double *s_eigs    = new double[nstate];
   double *scr1      = new double[3*nstate];        
   double *scr2      = new double[3*nstate];


//==========================================================================
// II. Diagonalize S using rs_ FORTRAN diagonalization routine

  int ifound = 0;
  int ierr   = 0;

  //----------------------------------------------------------------------
  // Use LAPACK : Captain Jack is Happy.
#ifdef _USE_LAPACK_
   ifound ++;
   for(int i = 1; i <= nstate; i++){
   for(int j = 1; j <= i; j++){
     int ind  = (i-1) + (j-1)*nstate;
     int ind2 = (i-1) + (j-1)*(2*nstate-j)/2;
     scr_mat1[ind2] = S[ind];
   }}//endfor
   char Vstuff    ='V';
   char Lstuff    ='L';
   DSPEV(&Vstuff,&Lstuff,&nstate,scr_mat1,s_eigs,umat,&nstate,scr1,&ierr);
#endif


   if(ifound!=1 || ierr != 0){
     PRINTF("@@@@@@@@@@@@@@@@@@@@_error_@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n");
     PRINTF("Error trying to diagonalize S : %d %d\n",ifound,ierr);
     PRINTF("@@@@@@@@@@@@@@@@@@@@_error_@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n");
     EXIT(1);
   }//endif

//==========================================================================
// III. Compute inverse square root of eigenvalues:  Occupation numbers 
//      are HACKED!!!!!

  //----------------------------------------------------------------------
  // A) Construct diagonal matrix using eigenvalues : sqrt(2/lamba)

   for(int i = 0; i < nstate; i++){s_eigs[i] = sqrt(2.0/s_eigs[i]);}
   memset(scr_mat1,0,sizeof(double)*nstate_sq);
   for(int i = 0; i < nstate; i++){
     int ind = i*nstate+i;
     scr_mat1[ind]=s_eigs[i];
   }/* endfor */

  //------------------------------------------------------------------------
  // B) Transform matrix back to original representation using eigenvectors

   double alpha = 1.0; double beta = 0.0;
   int itransp  = 0;   int inorm   = 1;

   matmul(scr_mat1,&nstate,&inorm,umat,&nstate,&itransp,scr_mat2,
             &nstate,&nstate,&nstate,&nstate,&alpha,&beta);
   matmul(umat,&nstate,&inorm,scr_mat2,&nstate,&inorm,T,
             &nstate,&nstate,&nstate,&nstate,&alpha,&beta);


//============================================================================
// IV) Free allocated temporary memory

   delete [] umat;
   delete [] scr_mat1;
   delete [] scr_mat2;
   delete [] s_eigs;
   delete [] scr1;
   delete [] scr2;

   cputime(&cpu2);
   PRINTF("nstate %d : cpu time diag : %g\n\n",nstate,cpu2-cpu1);

//============================================================================
  } /* End function */
//============================================================================



//============================================================================
//cccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccc
//============================================================================
// Set Tmax to the Unit matrix : remove cputime overhead of diag to test
//                               parallel performance
//============================================================================
void get_unit_Tmat(double *Tunit,int nstate){
   int nstate_sq = nstate*nstate;
   memset(Tunit,0,nstate_sq*sizeof(double));
   for(int i=0;i<nstate;i++){int ind = i+i*nstate;Tunit[ind] = 1.0;}
}
//============================================================================



//============================================================================
//cccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccc
//============================================================================
// Schulz iteration for inverse sqrt root : quadratic convergence!
//============================================================================

void get_iter_Tmat(double *S,double *Titer,int nstate)

//============================================================================
  {//begin routine
//============================================================================
// I) Get some scratch

   double cpu1,cpu2; 
   cputime(&cpu1);

   int nstate_sq     = nstate*nstate;
   double *scr_mat1  = new double[nstate_sq];
   double *scr_mat2  = new double[nstate_sq];
   double *scr_mat3  = new double[nstate_sq];

//============================================================================
// II) Set up CUBLAS context

    // cudaError_t cudaStat;           // cudaMalloc status 
    // cublasStatus_t stat;            // CUBLAS functions status 
    hipblasHandle_t handle;              // CUBLAS context

//============================================================================
// III) Allocate memory on the device
    double *d_Titer, *d_mat1, *d_mat2, *d_mat3;
    hipMalloc(&d_Titer, nstate_sq*sizeof(double));
    hipMalloc(&d_mat1, nstate_sq*sizeof(double));          
    hipMalloc(&d_mat2, nstate_sq*sizeof(double));          
    hipMalloc(&d_mat3, nstate_sq*sizeof(double));         

    hipblasCreate(&handle);                           // initialize CUBLAS context

//============================================================================
// IV) Schulz iteration

  //--------------------------------------------------------------------
  // A) Initialize scr_mat1 and Titer on host

    // scr_mat1 = S/2
    for(int i=0;i<nstate_sq;i++){scr_mat1[i] = S[i]/2.0;}
    // Titer = I = unit matrix
    memset(Titer,0,nstate_sq*sizeof(double));
    for(int i=0;i<nstate;i++){int ind = i+i*nstate;Titer[ind] = 1.0;}


    //--------------------------------------------------------------------
    // B) Initailize d_mat1 and d_Titer on device
    hipblasSetMatrix(nstate, nstate, sizeof(double), scr_mat1, nstate, d_mat1, nstate);       
    hipblasSetMatrix(nstate, nstate, sizeof(double), Titer, nstate, d_Titer, nstate);
    //cublasSetMatrix(m, n, sizeof(double), Z, m, d_Z, m);

    //--------------------------------------------------------------------
    // C) Iterate

    int iter        = 0;
    double tol_now  = 1.0;
    while (tol_now > 1.0e-15 && iter<10){

        iter++;
        //--------------------------------
        // scr_mat2 =  3*I - Titer*scr_mat1 
        int itransp  = 0;    int inorm    = 1;
        double alpha = -1.0; double beta  = 1.0;
        memset(scr_mat2,0,nstate_sq*sizeof(double));
        for(int i=0;i<nstate;i++){int ind = i+i*nstate;scr_mat2[ind]=3.0;}
        hipblasSetMatrix(nstate, nstate, sizeof(double), scr_mat2, nstate, d_mat2, nstate);
        device_matmul(d_Titer,&nstate,&inorm,d_mat1,&nstate,&itransp,d_mat2,
            &nstate,&nstate,&nstate,&nstate,&alpha,&beta,handle);
        //--------------------------------
        // scr_mat1 = 0.5*scr_mat1*scr_mat2 = 0.5*scr_mat3*scr_mat2
        alpha = 0.5;  beta  = 0.0;
        hipMemcpy(d_mat3,d_mat1,nstate_sq*sizeof(double),hipMemcpyDeviceToDevice);
        device_matmul(d_mat3,&nstate,&inorm,d_mat2,&nstate,&itransp,d_mat1,
            &nstate,&nstate,&nstate,&nstate,&alpha,&beta,handle);
        //--------------------------------
        // Titer = 0.5*scr_mat2*Titer = 0.5*scr_mat2*scr_mat3
        alpha = 0.5;  beta  = 0.0;
        hipMemcpy(d_mat3,d_Titer,nstate_sq*sizeof(double),hipMemcpyDeviceToDevice);
        device_matmul(d_mat2,&nstate,&inorm,d_mat3,&nstate,&itransp,d_Titer,
            &nstate,&nstate,&nstate,&nstate,&alpha,&beta,handle);
        //--------------------------------
        // tolerence check
        
	hipblasGetMatrix(nstate, nstate, sizeof(double), d_mat3, nstate, scr_mat3, nstate);
        hipblasGetMatrix(nstate, nstate, sizeof(double), d_Titer, nstate, Titer, nstate);
        tol_now = 0.0;
        for(int i=0;i<nstate_sq;i++){
            double tmp=scr_mat3[i]-Titer[i];
            tol_now += tmp*tmp;
        }//endfor
        tol_now /= ((double)nstate_sq);
        tol_now = sqrt(tol_now);
        PRINTF("iter %d : tol %g\n",iter,tol_now);
     
    }//endwhile

    if(tol_now>1.0e-15){
        PRINTF("@@@@@@@@@@@@@@@@@@@@_error_@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n");
        PRINTF("Iterative computation of S^{-1/2} failed\n");
        PRINTF("@@@@@@@@@@@@@@@@@@@@_error_@@@@@@@@@@@@@@@@@@@@@@@@@@@@\n");
        EXIT(1);
    }//endif

/*==========================================================================*/
// V) Copy the result back to the host
    hipblasGetMatrix(nstate, nstate, sizeof(double), d_Titer, nstate, Titer, nstate);
    // cublasGetMatrix(m, n, sizeof(double), d_Z, m, Z, m);

/*==========================================================================*/
// VI) Clean up device
    hipFree(d_Titer);
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_mat3);
    hipblasDestroy(handle);

// VII) Clean up host

    delete [] scr_mat1;
    delete [] scr_mat2;
    delete [] scr_mat3;
    
    cputime(&cpu2);
    PRINTF("nstate %d : cpu time iter : %g\n\n",nstate,cpu2-cpu1);

}//end routine
/*==========================================================================*/



/*==========================================================================*/
/*cccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccccc*/
/*==========================================================================*/
/* subroutine to time processes */
/*==========================================================================*/

void cputime(double *time)

/*==========================================================================*/
{
  int itime;
  static double to=0.,tn=0.;

  itime = clock();
  tn = (double)((double)itime/(double)CLOCKS_PER_SEC_C);
  *time = tn;
  if(tn >= 0 && to >= 0){*time=tn;}
  if(tn < 0  && to >= 0){*time=MAXTIME*2.0+tn;}
  if(tn >= 0 && to <  0){*time=tn+MAXTIME;}
  if(tn <  0 && to <  0){*time=MAXTIME+tn;}

  to = tn;
}
/*==========================================================================*/
